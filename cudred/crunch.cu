#include <hip/hip_runtime.h>

void calculateSumCombinations(float* input1, float* input2, float* output, int N)
{
    // Declare device memory pointers
    float *d_input1, *d_input2, *d_output;
    
    // Allocate memory on the GPU
    hipMalloc((void**)&d_input1, N * sizeof(float));
    hipMalloc((void**)&d_input2, N * sizeof(float));
    hipMalloc((void**)&d_output, N * N * sizeof(float));
    
    // Copy input data from host to device
    hipMemcpy(d_input1, input1, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input2, input2, N * sizeof(float), hipMemcpyHostToDevice);
    
    // Define the block and grid dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
    
    // Launch the kernel function
    sumCombinations<<<gridSize, blockSize>>>(d_input1, d_input2, d_output, N);
    
    // Wait for the GPU computation to complete
    hipDeviceSynchronize();
    
    // Copy the result from device to host
    hipMemcpy(output, d_output, N * N * sizeof(float), hipMemcpyDeviceToHost);
    
    // Free the GPU memory
    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);
}